#include <hip/hip_runtime.h>

__global__ void calx_kernel(double* xens, double* zens_wrap, double* a, int ensemble_size, int model_size, int ss2, int smooth_steps) 
{
    int iens = blockIdx.x * blockDim.x + threadIdx.x;

    if (iens >= ensemble_size)
    {
        return;
    }

    double *x = xens + iens * model_size;
    double *zwrap = zens_wrap + iens * model_size;

    for (int i = ss2; i < ss2 + model_size; i++)
    {
        x[i - ss2] = a[0] * zwrap[i + 1 - (- smooth_steps)] / 2.00;
        for (int j = -smooth_steps + 1; j < smooth_steps; j++)
        {
            x[i - ss2] = x[i - ss2] + a[j + smooth_steps] * zwrap[i + 1 - j];
        }
        x[i - ss2] = x[i - ss2] + a[2 * smooth_steps] * zwrap[i + 1 - smooth_steps] / 2.00;
    }
}


__global__ void calw(double* wxens, double* xens_wrap, int ensemble_size, int model_size, int K, int K4, int H) 
{
    int iens = blockIdx.x * blockDim.x + threadIdx.x;

    if (iens >= ensemble_size)
    {
        return;
    }

    double *wx = wxens + iens * model_size;
    double *xwrap = xens_wrap + iens * model_size;

    for (int i = K4; i < K4 + model_size; i++)
    {
        wx[i] = xwrap[i - (-H)] / 2.00;
        for (int j = -H + 1; j < H; j++)
            wx[i] = wx[i] + xwrap[i - j];

        wx[i] = wx[i] + xwrap[i - H] / 2.00;
        wx[i] = wx[i] / K;
    }
}


#ifdef __cplusplus
extern "C"
#endif

void run_calx_kernel(double* xens, double* zens_wrap, double* a, int ensemble_size, int model_size, int ss2, int smooth_steps)
{
    int block_size = 1024;
    int grid_size = (ensemble_size + block_size - 1) / block_size;

    calx_kernel<<<grid_size, block_size>>>(xens, zens_wrap, a, ensemble_size, model_size, ss2, smooth_steps);
}


void run_calw_kernel(double* wxens, double* xens_wrap, int ensemble_size, int model_size, int K, int K4, int H)
{
    int block_size = 1024;
    int grid_size = (ensemble_size + block_size - 1) / block_size;

    calw<<<grid_size, block_size>>>(wxens, xens_wrap, ensemble_size, model_size, K, K4, H);
}
